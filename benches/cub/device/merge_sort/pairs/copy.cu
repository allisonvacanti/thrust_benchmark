#include <nvbench/nvbench.cuh>

#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/sequence.h>
#include <thrust/shuffle.h>

#include <cub/device/device_merge_sort.cuh>

class less_comparator
{
public:
  template <typename T>
  __device__ bool operator()(T i, T j) noexcept
  {
    return i < j;
  }
};

template <typename KeyType, typename ValueType>
void custom_less(nvbench::state &state, nvbench::type_list<KeyType, ValueType>)
{
  const auto elements = static_cast<std::size_t>(state.get_int64("Elements"));

  thrust::device_vector<KeyType> keys_input(elements);
  thrust::device_vector<ValueType> values_input(elements);

  thrust::device_vector<KeyType> keys_output(elements);
  thrust::device_vector<ValueType> values_output(elements);

  thrust::sequence(keys_input.begin(), keys_input.end());

  thrust::default_random_engine rng;
  thrust::shuffle(keys_input.begin(), keys_input.end(), rng);

  state.add_element_count(elements);

  size_t temp_size = 0;
  hipcub::DeviceMergeSort::SortPairsCopy(
    nullptr,
    temp_size,
    thrust::raw_pointer_cast(keys_input.data()),
    thrust::raw_pointer_cast(values_input.data()),
    thrust::raw_pointer_cast(keys_output.data()),
    thrust::raw_pointer_cast(values_output.data()),
    elements,
    less_comparator());

  thrust::device_vector<char> tmp(temp_size);

  state.exec([&](nvbench::launch &launch) {
    NVBENCH_CUDA_CALL(hipcub::DeviceMergeSort::SortPairsCopy(
      thrust::raw_pointer_cast(tmp.data()),
      temp_size,
      thrust::raw_pointer_cast(keys_input.data()),
      thrust::raw_pointer_cast(values_input.data()),
      thrust::raw_pointer_cast(keys_output.data()),
      thrust::raw_pointer_cast(values_output.data()),
      elements,
      less_comparator(),
      launch.get_stream()));
  });
}
using value_types = nvbench::type_list<nvbench::uint8_t,
                                       nvbench::uint16_t,
                                       nvbench::uint32_t,
                                       nvbench::uint64_t,
                                       nvbench::float32_t,
                                       nvbench::float64_t>;

using key_types = nvbench::type_list<nvbench::uint8_t, nvbench::uint16_t>;

NVBENCH_BENCH_TYPES(custom_less, NVBENCH_TYPE_AXES(key_types, value_types))
  .set_name("hipcub::DeviceMergeSort::SortPairsCopy<custom_less> (random)")
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 27, 2));
